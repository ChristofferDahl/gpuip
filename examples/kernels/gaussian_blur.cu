#include "hip/hip_runtime.h"
__device__ float4 read(const unsigned short * in_half, int x, int y, int w)
{
    return make_float4(__half2float(in_half[4 * (x + y * w) + 0]),
                       __half2float(in_half[4 * (x + y * w) + 1]),
                       __half2float(in_half[4 * (x + y * w) + 2]),
                       __half2float(in_half[4 * (x + y * w) + 3]));
}

__device__ float weight(int i, int j, int x, int y, float invdx2)
{
    return exp(-invdx2*((i-x)*(i-x) + (j-y)*(j-y)));
}

__global__ void
gaussian_blur(const unsigned short * in_half,
              unsigned short * out_half,
              const int n,
              const int width,
              const int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // array index
    const int idx = x + width * y;

    // inside image bounds check
    if (x >= width || y >= height) {
        return;
    }

    // kernel code
    float4 out = make_float4(0, 0, 0, 0);
    const float invdx2 = 1.0/(width*width);
    float totWeight = 0;
    float w;
    for(int j = y - n; j <= y + n; ++j)  {
        for(int i = x - n; i <= x + n; ++i) {
            if (i>=0 && j>= 0 && i < width && j < height) {
                w = weight(i, j, x, y, invdx2);
                out += w * read(in_half, i, j, width);
                totWeight += w;
            }
        }
    }
    out /= totWeight;

    // float to half conversion
    out_half[4 * idx + 0] = __float2half_rn(out.x);
    out_half[4 * idx + 1] = __float2half_rn(out.y);
    out_half[4 * idx + 2] = __float2half_rn(out.z);
    out_half[4 * idx + 3] = __float2half_rn(out.w);
}
